#include "hipblas.h"
#include "hip/hip_runtime.h"
namespace nvinfer1
{
namespace plugin
{
__global__ void DoSelect(float const* input, float const* onehot, float* output)
{
    int cur_index = threadIdx.x;
    if (onehot[cur_index] > 0.5f)
    {
        // int index1 = cur_index / 4;
        // int index2 = cur_index % 4;
        // output[cur_index] = input[cur_index * 4];
        // output[cur_index + 1] = input[cur_index * 4 + 1];
        // output[cur_index + 2] = input[cur_index * 4 + 2];
        // output[cur_index + 3] = input[cur_index * 4 + 3];
        

        int out_index = (cur_index / 4) *4;
        output[out_index] = input[cur_index * 4];
     
        output[out_index + 1] = input[cur_index * 4 + 1];
        
        output[out_index + 2] = input[cur_index * 4 + 2];
       
        output[out_index + 3] = input[cur_index * 4 + 3];
        // output[out_index] = cur_index;
        
        // printf("%d  \n", cur_index);
    }
    // output[0] = 1.11f;
}
void RealSelect(float const* input, float const* onehot, float* output,int onehot_count)
{
    DoSelect<<<1, onehot_count>>>(input, onehot, output);
}
} // namespace plugin
} // namespace nvinfer1