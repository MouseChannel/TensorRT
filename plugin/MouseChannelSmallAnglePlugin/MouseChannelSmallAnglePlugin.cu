#include "hipblas.h"
#include "hip/hip_runtime.h"
namespace nvinfer1
{
namespace plugin
{
__global__ void DoHandle(float* sin_half_angles_over_angles, float const* small_angles, const float* angles,float* output)
{
    int cur_index = threadIdx.x;
    if (small_angles[cur_index] > 0.5f)
    {
        output[cur_index] = 0.5f - (angles[cur_index] * angles[cur_index]) / 48.f;
    }
    else
    {
        output[cur_index] = sin(angles[cur_index] / 2) / angles[cur_index];
    }
    // sin_half_angles_over_angles[cur_index] = cur_index;
    // output[0] = 1.11f;
}
void RealHandle(float* sin_half_angles_over_angles, float const* small_angles, const float* angles,float* output, int onehot_count)
{
    DoHandle<<<1, onehot_count>>>(sin_half_angles_over_angles, small_angles, angles,  output);
}
} // namespace plugin
} // namespace nvinfer1